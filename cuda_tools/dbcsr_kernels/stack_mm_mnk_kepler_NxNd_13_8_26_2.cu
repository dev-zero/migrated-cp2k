// DBCSR_KERNEL datatype=dbcsr_type_real_8, homogeneous_only=True, m=13, n=8, k=26
#include "dbcsr_kernel.h"
#include "dbcsr_generic_kernel.h"

int
launch_stack_mm_mnk_kepler_NxNd_13_8_26_2 (int *param_stack, int stack_size,
					   hipStream_t stream, int m_max,
					   int n_max, int k_max,
					   double *a_data, double *b_data,
					   double *c_data)
{
  int shared_size = 0;
  int careful = (stack_size / GROUPING);
  int nruns = stack_size - careful * GROUPING;
  stack_mm_mnk_kepler_NxNd < 13, 8, 26,
    2 > <<<((stack_size + GROUPING - 1) / GROUPING), 192, shared_size,
    stream >>> (param_stack, careful, nruns, m_max, n_max, k_max, a_data,
		b_data, c_data);
  return (0);
}
